#include "hip/hip_runtime.h"
#include <cstdlib>
#include <ctime>

#include <algorithm>
#include <iostream>
using namespace std;

__device__ void scan(unsigned *g_odata, unsigned *g_idata, int n) {
	extern __shared__ unsigned temp[];
	int thid = threadIdx.x;
	int pout = 0, pin = 1;

	temp[pin*n + thid] = 0;

	temp[pout * n + thid] = (thid > 0) ? g_idata[thid - 1] : 0;
	__syncthreads();
	for (int offset = 1; offset < n; offset *= 2) {
		if(offset == 1) break;
		pout = 1 - pout;
		pin = 1 - pout;
		if (thid >= offset)
			temp[pout * n + thid] += temp[pin * n + thid - offset];
		else
			temp[pout * n + thid] = temp[pin * n + thid];
		__syncthreads();
	}
	g_odata[thid] = temp[pout * n + thid];
}

__global__ void radix_sort(unsigned* numbers, unsigned length) {
	scan(numbers, numbers, length);
}

void print(unsigned* a, unsigned length) {
	cout << "[";
	for (unsigned i = 0; i < length; ++i) {
		cout << a[i] << " ";
	}
	cout << "]" << endl;
}

void fill(unsigned* a, unsigned length) {
	generate_n(a, length, []() {return rand()%4;});
}

int main(int argc, char **argv) {
	srand(time(nullptr));
	int devID = 0;
	hipError_t error;
	hipDeviceProp_t deviceProp;
	error = hipGetDevice(&devID);

	error = hipGetDeviceProperties(&deviceProp, devID);

	if (deviceProp.computeMode == hipComputeModeProhibited) {
		return 1;
	}

	if (error != hipSuccess) {

	}
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	constexpr unsigned int block_size = 4;
	constexpr unsigned int size = block_size * 1;

	unsigned *h_numbers = new unsigned[size];
//	fill(h_numbers, size);
	h_numbers[0] = 0;
	h_numbers[1] = 1;
	h_numbers[2] = 1;
	h_numbers[3] = 0;
	print(h_numbers, size);

	unsigned *d_numbers;
	hipMalloc((void**) &d_numbers, size * sizeof(*d_numbers));
	hipMemcpy(d_numbers, h_numbers, size * sizeof(*d_numbers),
			hipMemcpyHostToDevice);
	dim3 threadsPerBlock { block_size };
	dim3 blocksPerGrid { size / threadsPerBlock.x };

	hipEventRecord(start);
	radix_sort<<< blocksPerGrid,threadsPerBlock, 2*size >>> (d_numbers,size);
	hipEventRecord(stop);

	hipMemcpy(h_numbers, d_numbers, size * sizeof(*d_numbers),
			hipMemcpyDeviceToHost);

	hipEventSynchronize(stop);

	print(h_numbers, size);
	hipFree(d_numbers);
	delete[] h_numbers;

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	std::cout << "radixsort (" << milliseconds << " ms)" << std::endl;
	return 0;
}

