#include "hip/hip_runtime.h"
#include <cstdlib>
#include <ctime>
#include <climits>

#include <algorithm>
#include <functional>
#include <iostream>
using namespace std;

/*========* CudaArray *========*/

template<typename T>
class CudaArray {
public:
	CudaArray(int size) :
			size_ { size } {
		host_data = (T*) malloc(sizeof(T) * size);
		hipMalloc(&device_data, sizeof(T) * size);
	}
	~CudaArray() {
		free(host_data);
		hipFree(device_data);
	}
	int size() const {
		return size_;
	}
	T* host() {
		return host_data;
	}
	const T* host() const {
		return host_data;
	}
	T*& device() {
		return device_data;
	}
	const T* device() const {
		return device_data;
	}

	void copyToDevice() {
		hipMemcpy(device_data, host_data, sizeof(T) * size_,
				hipMemcpyHostToDevice);
	}
	void copyFromDevice() {
		hipMemcpy(host_data, device_data, sizeof(T) * size_,
				hipMemcpyDeviceToHost);
	}

private:
	int size_;
	T* host_data;
	T* device_data;
};

template<typename T>
void print(CudaArray<T>& a) {
	cout << "[ ";
	for (int i = 0; i < a.size(); ++i) {
		cout << a.host()[i] << " ";
	}
	cout << "]" << endl;
}

/*========* ArrayUtils *========*/

template<typename T, typename Fun>
void for_each(T *a, int length, Fun f) {
	for (int i = 0; i < length; ++i) {
		f(a[i], i);
	}
}

template<typename T>
void print(T *a, int length) {
	for_each(a, length, [](T& x, int i) {
		cout << x << " ";
	});
	cout << endl;
}

template<typename T, size_t N>
void print(T (&a)[N]) {
	print(a, N);
}

/*========* compile-time utils *========*/

constexpr unsigned ilog2_impl(unsigned x, unsigned bit) {
	return (1 << bit) & x ? bit : ilog2_impl(x, bit - 1);
}

constexpr unsigned ilog2(unsigned x) {
	return ilog2_impl(x, sizeof(unsigned) * CHAR_BIT - 1u);
}

constexpr unsigned next_multiple(unsigned multiple, unsigned x) {
	return x + (x % multiple == 0 ? 0 : (multiple - x % multiple));
}

/*========* device code *========*/

//__host__ __device__ unsigned id(unsigned x) {
//	return x;
//}
//
//__host__ __device__ unsigned is_zero(unsigned bit, unsigned x) {
//	return ((1 << bit) & ~x) >> bit;
//}
//
//template<typename Fun>
//__device__ void transform_scan(unsigned *idata, unsigned *odata, int n,
//		Fun fun) {
//	__shared__ unsigned temp[THREADS_PER_BLOCK * sizeof(unsigned)];
//	int thid = threadIdx.x;
//	int offset = 1;
//	temp[2 * thid] = fun(idata[2 * thid]);
//	temp[2 * thid + 1] = fun(idata[2 * thid + 1]);
//	for (int d = n >> 1; d > 0; d >>= 1) {
//		__syncthreads();
//		if (thid < d) {
//			int ai = offset * (2 * thid + 1) - 1;
//			int bi = offset * (2 * thid + 2) - 1;
//			temp[bi] += temp[ai];
//		}
//		offset *= 2;
//	}
//	if (thid == 0) {
//		temp[n - 1] = 0;
//	}
//	for (int d = 1; d < n; d *= 2) {
//		offset >>= 1;
//		__syncthreads();
//		if (thid < d) {
//			int ai = offset * (2 * thid + 1) - 1;
//			int bi = offset * (2 * thid + 2) - 1;
//			unsigned t = temp[ai];
//			temp[ai] = temp[bi];
//			temp[bi] += t;
//		}
//	}
//	__syncthreads();
//	odata[2 * thid] = temp[2 * thid];
//	odata[2 * thid + 1] = temp[2 * thid + 1];
//}
//
//__global__ void transform_scan_all(unsigned *idata, unsigned *odata, int n,
//		unsigned bit) {
//	int offset = 2 * blockDim.x * blockIdx.x;
//	transform_scan(idata + offset, odata + offset, 2 * blockDim.x,
//			[bit](unsigned x) {
//				return is_zero(bit,x);
//			});
//}
//__global__ void scan_all(unsigned *idata, unsigned *odata, int n) {
//	int offset = 2 * blockDim.x * blockIdx.x;
//	transform_scan(idata + offset, odata + offset, 2 * blockDim.x, id);
//}
//
//__device__ unsigned *scan_buffer;
//__global__ void radix_sort(unsigned *numbers, int n, unsigned bit) {
//	int idx = threadIdx.x + blockDim.x * blockIdx.x;
//	if (idx == 0) {
//		scan_buffer = new unsigned[n];
//	}
//	int offset = 2 * blockDim.x * blockIdx.x;
//	__syncthreads();
//	transform_scan(numbers + offset, scan_buffer + offset, 2 * blockDim.x,
//			[bit](unsigned x) {
//				return is_zero(bit, x);
//			});
//	__syncthreads();
//	const unsigned zeros = scan_buffer[n - 1] + is_zero(numbers[n - 1], bit);
//	const unsigned tmp1 = numbers[2 * idx];
//	const unsigned tmp2 = numbers[2 * idx + 1];
//
//	__syncthreads();
//	const auto new_index = [zeros,bit](unsigned x, unsigned i) {
//		return is_zero(x,bit)
//		? scan_buffer[i]
//		: i - scan_buffer[i] + zeros;
//	};
//	__syncthreads();
//	numbers[new_index(tmp1, 2 * idx)] = tmp1;
//	numbers[new_index(tmp2, 2 * idx + 1)] = tmp2;
//
//	__syncthreads();
//	if (idx == 0) {
//		delete[] scan_buffer;
//	}
//}
__host__ __device__ unsigned is_zero(unsigned bit, unsigned x) {
	return ((1 << bit) & ~x) >> bit;
}
__device__ void block_scan(unsigned *data, int n, unsigned *sum) {
	extern __shared__ unsigned temp[];
	int thid = threadIdx.x;
	int offset = 1;
	temp[2 * thid] = data[2 * thid];
	temp[2 * thid + 1] = data[2 * thid + 1];
	for (int d = n >> 1; d > 0; d >>= 1) {
		__syncthreads();
		if (thid < d) {
			int ai = offset * (2 * thid + 1) - 1;
			int bi = offset * (2 * thid + 2) - 1;
			temp[bi] += temp[ai];
		}
		offset *= 2;
	}
	if (thid == 0) {
		*sum = temp[n - 1];
		temp[n - 1] = 0;
	}
	for (int d = 1; d < n; d *= 2) {
		offset >>= 1;
		__syncthreads();
		if (thid < d) {
			int ai = offset * (2 * thid + 1) - 1;
			int bi = offset * (2 * thid + 2) - 1;
			unsigned t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}
	__syncthreads();
	data[2 * thid] = temp[2 * thid];
	data[2 * thid + 1] = temp[2 * thid + 1];
}
__global__ void partial_scan(unsigned *data, int n, unsigned *sums) {
	int offset = 2 * blockDim.x * blockIdx.x;
	block_scan(data + offset, 2 * blockDim.x, sums + blockIdx.x);
}

__global__ void d_fill_n(unsigned *data, unsigned n, unsigned value) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < n) {
		data[idx] = value;
	}
}

__global__ void add_sums(unsigned *numbers, unsigned *block_sums) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	numbers[2 * idx] += block_sums[blockIdx.x];
	numbers[2 * idx + 1] += block_sums[blockIdx.x];
}

__global__ void d_transform(unsigned *numbers, unsigned *new_numbers,
		unsigned bit) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	new_numbers[idx] = is_zero(bit, numbers[idx]);
}
__global__ void rearange(unsigned *in_numbers, unsigned *out_numbers,
		unsigned *scan, unsigned count, unsigned bit) {
//	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	auto zeros = scan[count - 1] + is_zero(bit, in_numbers[count - 1]);
	int blockOffset = (2 * blockDim.x) * blockIdx.x;

	int myindex1 = blockOffset + 2 * threadIdx.x;
	int myindex2 = blockOffset + 2 * threadIdx.x + 1;

	int val1 = in_numbers[myindex1];
	int val2 = in_numbers[myindex2];

	int index1 =
			is_zero(bit, val1) ?
					scan[myindex1] : myindex1 - scan[myindex1] + zeros;
	int index2 =
			is_zero(bit, val2) ?
					scan[myindex2] : myindex2 - scan[myindex2] + zeros;

	out_numbers[index1] = val1;
	out_numbers[index2] = val2;
}

/*========* host code *========*/

void scan(unsigned *d_numbers, const unsigned count,
		const unsigned block_size) {
	const unsigned count_filled = next_multiple(block_size, count);

	const unsigned block_dim = block_size / 2;
	const unsigned grid_dim = count_filled / block_dim;

	unsigned *d_block_sums;
	hipMalloc(&d_block_sums, (grid_dim * 2) * sizeof(unsigned));

	d_fill_n<<<grid_dim, block_dim>>>(d_numbers + count, count_filled - count, 0);

	partial_scan<<<grid_dim, block_dim, 2*block_size*sizeof(unsigned)>>>(d_numbers, count_filled, d_block_sums);

	if (count_filled > block_size) {
		scan(d_block_sums, grid_dim, block_size);
		add_sums<<<grid_dim, block_dim>>>(d_numbers, d_block_sums);
	}
	hipFree(d_block_sums);
}

void test_error(int str) {
	auto error = hipGetLastError();
	if (error != hipSuccess) {
		cout << str << ":" << hipGetErrorString(error) << endl;
	}
}

void foo() {
	srand(time(nullptr));
	constexpr auto BLOCK_SIZE = 4;
	constexpr auto N = 128*1024;


	static_assert((1u<<ilog2(BLOCK_SIZE)) == BLOCK_SIZE,"THREADS_PER_BLOCK muss eine 2er Potenz sein");
	constexpr auto N_FILLED = next_multiple(BLOCK_SIZE, N);
	const unsigned block_dim = BLOCK_SIZE / 2;
	const unsigned grid_dim = N_FILLED / BLOCK_SIZE;

	cout << "foooo:" <<(N == N_FILLED) << endl;

	CudaArray<unsigned> a { N_FILLED };
	std::generate_n(a.host(), a.size(), []() {return rand()%16;});
//	int x = N_FILLED;
//	std::generate_n(a.host(), a.size(), [x]() mutable {return x--;});
//	std::fill_n(a.host(), a.size(),0);
//	print(a);

	a.copyToDevice();

	CudaArray<unsigned> d_buffer { N_FILLED };
	CudaArray<unsigned> d_scan { N_FILLED };
//	unsigned *d_scan;
//	unsigned *d_buffer;
//	hipMalloc(&d_scan, N_FILLED * sizeof(unsigned));
//	hipMalloc(&d_buffer, N_FILLED * sizeof(unsigned));
	for (int bit = 0; bit < 32; ++bit) {
		d_transform<<<grid_dim,BLOCK_SIZE>>>(a.device(), d_scan.device(), bit);
		test_error(__LINE__);
		scan(d_scan.device(), N, BLOCK_SIZE);
		test_error(__LINE__);
		rearange<<<grid_dim,block_dim>>>(a.device(), d_buffer.device(), d_scan.device(), N_FILLED, bit);
		test_error(__LINE__);
		d_scan.copyFromDevice();
		print(d_scan);
//		d_buffer.copyFromDevice();
//		print(d_buffer);
//		break;
		unsigned *tmp = a.device();
		a.device() = d_buffer.device();
		d_buffer.device() = tmp;
	}
//	hipFree(d_scan);
//	hipFree(d_buffer);
	a.copyFromDevice();
//	print(a);
	cout << is_sorted(a.host(), a.host() + a.size()) << endl;
}

void test_partial_scan() {
	constexpr auto BLOCK_SIZE = 4;
	constexpr auto N = 16;

	static_assert((1u<<ilog2(BLOCK_SIZE)) == BLOCK_SIZE,"THREADS_PER_BLOCK muss eine 2er Potenz sein");
	constexpr auto N_FILLED = next_multiple(BLOCK_SIZE, N);

	CudaArray<unsigned> a { N_FILLED };
	CudaArray<unsigned> sums { ilog2(N_FILLED) };
	std::generate_n(a.host(), a.size(), []() {return rand()%4;});
	print(a);
	a.copyToDevice();
	partial_scan<<<N/BLOCK_SIZE,BLOCK_SIZE/2,BLOCK_SIZE>>>(a.device(), a.size(), sums.device());
	a.copyFromDevice();
	sums.copyFromDevice();

	print(a);
	print(sums);

	auto error = hipGetLastError();
	if (error != hipSuccess) {
		cout << hipGetErrorString(error) << endl;
	}
}

//void scan(unsigned *block_max_elem, unsigned n) {
//	if (n <= THREADS_PER_BLOCK)
//		return;
//	const auto N = next_multiple(THREADS_PER_BLOCK, n / THREADS_PER_BLOCK);
//	CudaArray<unsigned> max_elems { N };
//
//	for (int i = 0; i < N; ++i) {
//		max_elems.host()[i] = block_max_elem[(i + 1) * THREADS_PER_BLOCK - 1]
//				+ block_max;
//	}
//
//	max_elems.copyToDevice();
//	scan_all<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK/2>>>(max_elems.device(),max_elems.device(),max_elems.size());
//	max_elems.copyFromDevice();
//
//	scan(max_elems.host(), N);
//
//	for (int i = 0; i < N; ++i) {
//		for (int j = 0; j < THREADS_PER_BLOCK; ++j) {
//			block_max_elem[i * THREADS_PER_BLOCK + j] += max_elems.host()[i];
//		}
//	}
//}

//void foo() {
//	srand(time(nullptr));
//	CudaArray<unsigned> a { N_FILLED };
//	CudaArray<unsigned> block_scan { N_FILLED };
//	CudaArray<unsigned> block_max_elem { N_FILLED / THREADS_PER_BLOCK };
//
//	std::fill_n(a.host(), N, 0);
////	std::generate_n(a.host(), a.size(), []() {return rand()%256;});
//	unsigned x = 1;
////	std::generate_n(a.host(), N, [x]() mutable {return x++;});
//	std::fill_n(a.host() + N, N_FILLED - N, UINT_MAX);
////	a.host()[0] = 3; //0
////	a.host()[1] = 0; //1
////	a.host()[2] = 2; //1
////	a.host()[3] = 1; //0
////	a.host()[4] = 3; //0
////	a.host()[5] = 0; //1
////	a.host()[6] = 2; //1
////	a.host()[7] = 1; //0
//
//	print(a.host(), N);
//
//	a.copyToDevice();
//	for (unsigned bit = 0u; bit < 32; ++bit) {
//		transform_scan_all<<<N_FILLED/THREADS_PER_BLOCK,THREADS_PER_BLOCK/2>>>(a.device(),block_scan.device(),N_FILLED,bit);
//		block_scan.copyFromDevice();
//		scan(block_scan.host(),block_scan.size());
//		break;
//	}
//	a.copyFromDevice();
//
//	bool ok = true;
//	x = 1;
//	for (int i = 0; i < N; ++i) {
//		if (a.host()[i] != x++) {
//			ok = false;
//			cout << i << endl;
//			break;
//		}
//	}
//	cout << "ok: " << ok << endl;
//	auto error = hipGetLastError();
//	if (error != hipSuccess) {
//		cout << hipGetErrorString(error) << endl;
//	}
//	print(block_scan.host(), N);
//}

int main(int argc, char **argv) {
	int devID = 0;
	hipError_t error;
	hipDeviceProp_t deviceProp;
	error = hipGetDevice(&devID);

	error = hipGetDeviceProperties(&deviceProp, devID);

	if (deviceProp.computeMode == hipComputeModeProhibited) {
		return 1;
	}

	if (error != hipSuccess) {

	}
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	foo();

	hipEventRecord(stop);

	hipEventSynchronize(stop);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	cout << "time (" << milliseconds << " ms)" << endl;
	return 0;
}

