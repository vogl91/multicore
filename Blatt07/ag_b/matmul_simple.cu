#include "hip/hip_runtime.h"
#include <algorithm>
#include <random>
#include <iostream>
#include <iomanip>
#include <functional>

constexpr int BLOCK_SIZE = 32;

__global__ void matrix_mult(float* C, float* A, float* B, int size) {
	int bx = blockIdx.x;
	int by = blockIdx.y;

	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int i = by * blockDim.y + ty;
	int j = bx * blockDim.x + tx;

	float sum = 0.0;
#pragma unroll
	for (int k = 0; k < size; k++) {
		sum += A[i * size + k] * B[k * size + j];
	}

	C[i * size + j] = sum;
}

struct Matrix {
	Matrix(int size) :
			size { size }, data { new float[size * size] } {
	}
	~Matrix() {
		delete[] data;
	}
	const int size;
	float* data;
	float& get(int i, int j) {
		return data[i * size + j];
	}
};

void fill(Matrix& a) {
	const auto min = 0.0f;
	const auto max = 1.0f;
	std::random_device real_random;
	std::mt19937 random { real_random() };
	std::uniform_real_distribution<float> distribution { min, max };
	auto next_val = std::bind(distribution, random);
	std::generate(a.data, a.data + a.size * a.size, next_val);
}

bool is_equal(Matrix& a, Matrix& b) {
	const auto epsilon = 0.0001;
	if (a.size != b.size)
		return false;
	for (auto i = 0; i < a.size * a.size; ++i) {
		if (abs(a.data[i] - b.data[i]) > epsilon)
			return false;
	}
	return true;
}

bool is_correct(Matrix& c, Matrix& a, Matrix& b) {
	const auto epsilon = 0.0001;
	for (int i = 0; i < c.size; ++i) {
		for (int j = 0; j < c.size; ++j) {
			float sum = 0.0f;
			for (int k = 0; k < c.size; ++k) {
				sum += a.get(i, k) * b.get(k, j);
			}
			if (abs(sum - c.get(i, j) > epsilon))
				return false;
		}
	}
	return true;
}

int main(int argc, char **argv) {
	int devID = 0;

	hipError_t error;
	hipDeviceProp_t deviceProp;
	error = hipGetDevice(&devID);

	error = hipGetDeviceProperties(&deviceProp, devID);

	if (deviceProp.computeMode == hipComputeModeProhibited) {
		return 1;
	}

	if (error != hipSuccess) {

	}
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	{
		constexpr auto matrix_size = 1024;
//		constexpr auto matrix_size = 2048;
//		constexpr auto matrix_size = 4096;
//		constexpr auto matrix_size = 8192;
		Matrix h_A { matrix_size }, h_B { matrix_size }, h_C { matrix_size };
		fill(h_A);
		fill(h_B);

		float* d_A, *d_B, *d_C;

		hipEventRecord(start);
		const int size_in_bytes = matrix_size * matrix_size * sizeof(float);
		//make space for device matrix representation
		hipMalloc((void**) &d_A, size_in_bytes);
		hipMalloc((void**) &d_B, size_in_bytes);
		hipMalloc((void**) &d_C, size_in_bytes);

		//copy input matrix to device
		hipMemcpy(d_A, h_A.data, size_in_bytes, hipMemcpyHostToDevice);
		hipMemcpy(d_B, h_B.data, size_in_bytes, hipMemcpyHostToDevice);

		dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
		dim3 blocksPerGrid(matrix_size / BLOCK_SIZE, matrix_size / BLOCK_SIZE,
				1);
		matrix_mult<<<blocksPerGrid, threadsPerBlock>>>(d_C, d_A, d_B,
				matrix_size);

		//copy output matrix to host
		hipMemcpy(h_C.data, d_C, size_in_bytes, hipMemcpyDeviceToHost);
		hipFree(d_A);
		hipFree(d_B);
		hipFree(d_C);
		hipEventRecord(stop);

		hipEventSynchronize(stop);

		float milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start, stop);

		std::cout << "Matrixmultiplikation (" << milliseconds << " ms)"
				<< std::endl;

		std::cout << "is_correct:" << std::boolalpha
				<< is_correct(h_C, h_A, h_B) << std::endl;
	}

	return 0;
}

