#include "hip/hip_runtime.h"
#include <iostream>


__global__ void matrix_mult(){


}



int main(int argc, char **argv)
{
    int devID = 0;

    hipError_t error;
    hipDeviceProp_t deviceProp;
    error = hipGetDevice(&devID);

    error = hipGetDeviceProperties(&deviceProp, devID);

    if (deviceProp.computeMode == hipComputeModeProhibited)
    {
    	return 1;
    }

    if (error != hipSuccess)
    {

    }
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);


    hipEventRecord(start);		
    matrix_mult<<< 1,1 >>> ();
    hipEventRecord(stop); 

    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << "Matrixmultiplikation (" << milliseconds << " ms)" << std::endl;
    
    return 0;
}

